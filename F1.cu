#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include <unordered_set>
#include <unordered_map>
#include <random>






__global__ void getF1(float* TP, float* FP, float* FN, float* trueValuesByClass, int* noClasses, float* samplesPerClass, float* F1_Macro, float* F1_Weighted, int noTargetValues, int noIndividuals){
 int idx = (blockDim.x * blockDim.y * blockIdx.y * gridDim.x) + (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y + threadIdx.x);
 
 if (idx < *noClasses * noIndividuals){
    int individual;
    if (idx < *noClasses) individual = 0; else individual = ((idx - noTargetValues) / noTargetValues) + 1;
    atomicAdd(&F1_Macro[individual], ( TP[idx] / (TP[idx] + 0.5 * (FP[idx] + FN[idx])) ));
    atomicAdd(&F1_Weighted[individual], ( (samplesPerClass[idx - individual * (*noClasses)] / noTargetValues) * (TP[idx] / (TP[idx] + 0.5 * (FP[idx] + FN[idx]))) ));  
 }
   
}




__global__ void getTpFpFn(float* y_true, float* y_pred, int n, int m, int noClasses, float* y_trueEachClass, float* TP, float* FP, float* FN){
    int idx = (blockDim.x * blockDim.y * blockIdx.y * gridDim.x) + (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y + threadIdx.x);
    if (idx < m * n) {
        int individual;
        if (idx < n) individual = 0;
        else individual = ((idx - n) / n) + 1;
        for (int i = 0; i < noClasses; i++){
            printf("y_pred[%i]: %f != y_trueEachClass[%i]: %f && y_true[%i]: %f == y_trueEachClass[%i]: %f = %d\n", idx, y_pred[idx], i, y_trueEachClass[i], idx - individual * n, y_true[idx - individual * n], i, y_trueEachClass[i], (y_pred[idx] != y_trueEachClass[i] && y_true[idx - individual * n] == y_trueEachClass[i]));
            if(y_pred[idx] == y_trueEachClass[i] && y_true[idx - individual * n] == y_trueEachClass[i]) { atomicAdd(&TP[i + individual * noClasses], 1); }
            if(y_pred[idx] == y_trueEachClass[i] && y_true[idx - individual * n] != y_trueEachClass[i]) { atomicAdd(&FP[i + individual * noClasses], 1); }
            if(y_pred[idx] != y_trueEachClass[i] && y_true[idx - individual * n] == y_trueEachClass[i]) { atomicAdd(&FN[i + individual * noClasses], 1); }
        }
    }
}


__global__ void getNoClassesKernel(float* trueValues, int* noClasses, float* temp, int* count, int* i){
 if(trueValues[threadIdx.x] == 0) *noClasses = *noClasses + 1;
 atomicAdd(count, 0);
 if(trueValues[threadIdx.x] != 0) {
     *noClasses = *noClasses + 1;
     *temp = trueValues[threadIdx.x];
 }
 atomicAdd(count, 0);
 for ( ; *i < 1; *i = *i + 1)
 {
     if(trueValues[threadIdx.x] == *temp) trueValues[threadIdx.x] = 0;
     if(trueValues[threadIdx.x] != 0) {
         *noClasses = *noClasses + 1;
         *temp = trueValues[threadIdx.x];
         *i = *i - 1;
     }
     atomicAdd(count, 0);
 }
}

__global__ void getClasses(float* trueValues, int* noClasses, float* temp, int* count, float* trueValuesByClass, int* i){
 if(trueValues[threadIdx.x] == 0) { trueValuesByClass[0] = 0; *i = *i + 1; }
 atomicAdd(count, 0);
 for ( ; *i < *noClasses; *i = *i + 1)
 {
     if(trueValues[threadIdx.x] != 0) *temp = trueValues[threadIdx.x];
     atomicAdd(count, 0);
     trueValuesByClass[*i] = *temp;
     if(trueValues[threadIdx.x] == *temp) trueValues[threadIdx.x] = 0;
     atomicAdd(count, 0);
 }
}

__global__ void getSamplesPerClass(float* trueValues, int* noClasses, int* count, float* trueValuesByClass, float* samplesPerClass){
 for (int i = 0; i < *noClasses; i++) if(trueValues[threadIdx.x] == trueValuesByClass[i]) atomicAdd(&samplesPerClass[i], 1);
}

__global__ void getMacro(float* F1_Macro, int* noClasses, int noIndividuals){
    int idx = (blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * threadIdx.y + threadIdx.x);
    if (idx < noIndividuals)
    {
        F1_Macro[idx] /= *noClasses;
    }
}
void getVector(float* vector, int size){
 for (int i = 0; i < size; i++){
     if(i % 2 == 0) vector[i] = 1;
     else vector[i] = 2;
 } 
}

void getMatriz(float* matriz, int rows, int columns){
    for (int i = 0; i < rows; i++)
        for (int e = 0; e < columns; e++)
            matriz[i + e] = i;
    
}


void F1(float* y_pred, float* y_true, int widthTrue, int rowPred){
 float* y_trueDevice, * temp, * temp_d, * valuesByClass, * valuesByClasses_d, * samplesPerClass, * samplesPerClass_d, * y_predDevice, * TP, * FP, * FN, * TP_d, * FP_d, * FN_d;
 int* count, *noClasses, * i, * count_d, *noClasses_d, * i_d;




 temp = (float*)malloc(sizeof(float));
 count = (int*)malloc(sizeof(int));
 noClasses = (int*)malloc(sizeof(int));
 i = (int*)malloc(sizeof(int));




 *temp = 0;
 *count = 0;
 *noClasses = 0;
 *i = 0;


 hipMalloc((void**)&y_trueDevice, widthTrue * sizeof(float));
 hipMalloc((void**)&temp_d, sizeof(float));
 hipMalloc((void**)&count_d, sizeof(int));
 hipMalloc((void**)&noClasses_d, sizeof(int));
 hipMalloc((void**)&i_d, sizeof(int));




 hipMemcpy(y_trueDevice, y_true, widthTrue * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(temp_d, temp, sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(count_d, count, sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(noClasses_d, noClasses, sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(i_d, i, sizeof(int), hipMemcpyHostToDevice);




 getNoClassesKernel<<<1, widthTrue>>>(y_trueDevice, noClasses_d, temp_d, count_d, i_d);
 hipDeviceSynchronize();



 




 hipMemcpy(noClasses, noClasses_d, sizeof(int), hipMemcpyDeviceToHost);
 valuesByClass = (float*)malloc(*noClasses * sizeof(float));
 hipMalloc((void**)&valuesByClasses_d, *noClasses * sizeof(float));
 hipMemcpy(y_trueDevice, y_true, widthTrue * sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(i_d, i, sizeof(int), hipMemcpyHostToDevice);

 getClasses<<<1, widthTrue>>>(y_trueDevice, noClasses_d, temp_d, count_d, valuesByClasses_d, i_d);
 hipDeviceSynchronize();
 hipMemcpy(valuesByClass, valuesByClasses_d, *noClasses * sizeof(float), hipMemcpyDeviceToHost);








 samplesPerClass = (float*)malloc(*noClasses * sizeof(float));
 hipMalloc((void**)&samplesPerClass_d, *noClasses * sizeof(float));
 hipMemcpy(y_trueDevice, y_true, widthTrue * sizeof(float), hipMemcpyHostToDevice);




 getSamplesPerClass<<<1, widthTrue>>>(y_trueDevice, noClasses_d, count_d, valuesByClasses_d, samplesPerClass_d);
 hipDeviceSynchronize();
 hipMemcpy(samplesPerClass, samplesPerClass_d, *noClasses * sizeof(float), hipMemcpyDeviceToHost);








 TP = (float*)malloc(*noClasses * rowPred * sizeof(float));
 FP = (float*)malloc(*noClasses * rowPred * sizeof(float));
 FN = (float*)malloc(*noClasses * rowPred * sizeof(float));

dim3 block(32, 32);
dim3 grid((widthTrue + block.x - 1) / block.x, (rowPred + block.y - 1) / block.y);

 hipMalloc((void**)&TP_d, *noClasses * rowPred * sizeof(float));
 hipMalloc((void**)&FP_d, *noClasses * rowPred * sizeof(float));
 hipMalloc((void**)&FN_d, *noClasses * rowPred * sizeof(float));
 hipMalloc((void**)&y_predDevice, rowPred * widthTrue * sizeof(float));
 hipMemcpy(y_predDevice, y_pred, rowPred * widthTrue * sizeof(float), hipMemcpyHostToDevice);
 getTpFpFn<<<grid, block>>>(y_trueDevice, y_predDevice, widthTrue, rowPred, *noClasses, valuesByClasses_d, TP_d, FP_d, FN_d);
 hipDeviceSynchronize();


 hipMemcpy(TP, TP_d, *noClasses * sizeof(float), hipMemcpyDeviceToHost);
 hipMemcpy(FP, FP_d, *noClasses * sizeof(float), hipMemcpyDeviceToHost);
 hipMemcpy(FN, FN_d, *noClasses * sizeof(float), hipMemcpyDeviceToHost);
 
 
 
 float* F1_Macro, * F1_Macro_d;
 float* F1_Weighted, * F1_Weighted_d;
 F1_Macro = (float*)malloc(rowPred * sizeof(float));
 F1_Weighted = (float*)malloc(rowPred * sizeof(float));
 hipMalloc((void**)&F1_Macro_d, rowPred * sizeof(float));
 hipMalloc((void**)&F1_Weighted_d, rowPred * sizeof(float));

 dim3 grid1((*noClasses + block.x - 1) / block.x, (rowPred + block.y - 1) / block.y);

 getF1<<<grid, block>>>(TP_d, FP_d, FN_d, valuesByClasses_d, noClasses_d, samplesPerClass_d, F1_Macro_d, F1_Weighted_d, widthTrue, rowPred);
 hipDeviceSynchronize();
 hipMemcpy(F1_Macro, F1_Macro_d, rowPred * sizeof(float), hipMemcpyDeviceToHost);
 hipMemcpy(F1_Weighted, F1_Weighted_d, rowPred * sizeof(float), hipMemcpyDeviceToHost);
 
 dim3 grid2((rowPred + block.x - 1) / block.x);

 getMacro<<<grid2, block>>>(F1_Macro_d, noClasses_d, rowPred);
 hipDeviceSynchronize();
 hipMemcpy(F1_Macro, F1_Macro_d, rowPred * sizeof(float), hipMemcpyDeviceToHost);

 for (int i = 0; i < rowPred; i++)
 {
    printf("F1 MACRO[%i]: %f\n", i, F1_Macro[i]);
    printf("F1 WEIGHTED[%i]: %f\n", i, F1_Weighted[i]);
    for (int e = 0; e < *noClasses; e++)
    {
        printf("\nTP[clase: %i, individuo: %i]: %f\n", e, i, TP[i * e + e]);
        printf("\nFP[clase: %i, individuo: %i]: %f\n", e, i, TP[i * e + e]);
        printf("\nFN[clase: %i, individuo: %i]: %f\n", e, i, FN[i * e + e]);
    }
    
 }
 
 // printf("F1 MACRO: %f, TP_1: %f, FP_1: %f, FN_1: %f, TP_2: %f, FP_2: %f, FN_2: %f, TP_3: %f, FP_3: %f, FN_3: %f", *F1_Macro, TP[0], FP[0], FN[0], TP[1], FP[1], FN[1], TP[2], FP[2], FN[2]);
 // printf("\nF1 WEIGHTED: %f\n", *F1_Weighted);


}


int main(){ 
 // float* y_pred, * y_true, * y_trueEachClass;
 float y_pred[12] = {0, 2, 1, 0, 0, 1, 0, 2, 1, 0, 0, 1}; float y_true [7] = {0, 1, 2, 0, 1, 2};


 F1(y_pred, y_true, 6, 2);
}